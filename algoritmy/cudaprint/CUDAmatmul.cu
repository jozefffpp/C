#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define MAT_SIZE 4	//rozmer matice
#define BLOCK_SIZE 2	//rozmer bloku

//matice sú uložené v row-major formáte
//M(row, col) = *(M.elements + row * M.stride + col)
typedef struct {
  int width;
  int height;
  int stride; 
  float* elements;
} Matrix;

//prečítaj hodnotu prvku matice
__device__ float GetElement(const Matrix A, int row, int col)
{
  return A.elements[row * A.stride + col];
}

//zapíš hodnotu prvku matice
__device__ void SetElement(Matrix A, int row, int col,
                           float value)
{
  A.elements[row * A.stride + col] = value;
}

//načítaj blok matice
 __device__ Matrix GetSubMatrix(Matrix A, int row, int col) 
{
  Matrix Asub;
  Asub.width    = BLOCK_SIZE;
  Asub.height   = BLOCK_SIZE;
  Asub.stride   = A.stride;
  Asub.elements = &A.elements[A.stride * BLOCK_SIZE * row + BLOCK_SIZE * col];
  return Asub;
}

//kernel pre násobenie matíc
 __global__ void MatMulKernel(Matrix A, Matrix B, Matrix C)
{
  //identifikácia bloku
  int blockRow = blockIdx.y;
  int blockCol = blockIdx.x;

  //podmatica počitana v bloku
  Matrix Csub = GetSubMatrix(C, blockRow, blockCol);

  //hodnota prvku matice C počítaná vláknom
  float Cvalue = 0;

  //identifikácia vlákna
  int row = threadIdx.y;
  int col = threadIdx.x;

  //cyklus cez všetky podmatice 
  for (int m = 0; m < (A.width / BLOCK_SIZE); ++m) {

    //získanie parametrov podmatíc A a B
    Matrix Asub = GetSubMatrix(A, blockRow, m);
    Matrix Bsub = GetSubMatrix(B, m, blockCol);

    //zdieľaná pamäť na uloženie blokov Asub a Bsub
    __shared__ float As[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float Bs[BLOCK_SIZE][BLOCK_SIZE];

    //prenos podmatíc z globálnej do zdieľanej pamäte
    As[row][col] = GetElement(Asub, row, col);
    Bs[row][col] = GetElement(Bsub, row, col);

    //synchronizácia vlákien
    __syncthreads();

    //násobenie Asub a Bsub
    for (int e = 0; e < BLOCK_SIZE; ++e)
      Cvalue += As[row][e] * Bs[e][col];

    //synchronizácia vlákien
    __syncthreads();
  }

  //zápis výslednej hodnoty prvku
  SetElement(Csub, row, col, Cvalue);
}

//hostiteľská časť programu
void MatMul(const Matrix A, const Matrix B, Matrix C)
{
  //alokácia pamäte zariadenia GPGPU
  Matrix d_A;
  d_A.width = d_A.stride = A.width; d_A.height = A.height;
  size_t size = A.width * A.height * sizeof(float);
  hipMalloc(&d_A.elements, size);
  //kopírovanie údajov do GPGPU pamäte
  hipMemcpy(d_A.elements, A.elements, size, hipMemcpyHostToDevice);

  //alokácia pamäte zariadenia GPGPU
  Matrix d_B;
  d_B.width = d_B.stride = B.width; d_B.height = B.height;
  size = B.width * B.height * sizeof(float);
  hipMalloc(&d_B.elements, size);
  //kopírovanie údajov do GPGPU pamäte
  hipMemcpy(d_B.elements, B.elements, size, hipMemcpyHostToDevice);

  //alokácia pamäte zariadenia GPGPU
  Matrix d_C;
  d_C.width = d_C.stride = C.width; d_C.height = C.height;
  size = C.width * C.height * sizeof(float);
  hipMalloc(&d_C.elements, size);

  //volanie kernelu
  dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
  dim3 dimGrid(B.width / dimBlock.x, A.height / dimBlock.y);
  MatMulKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C);

  //synchronizácia - dokončenie kernelu
  hipDeviceSynchronize();

  //kopírovanie údajov z GPGPU pamäte
  hipMemcpy(C.elements, d_C.elements, size, hipMemcpyDeviceToHost);

  //uvoľnenie pamäte zariadenia GPGPU
  hipFree(d_A.elements);
  hipFree(d_B.elements);
  hipFree(d_C.elements);
}

int main()
{
  srand(time(NULL));
  //alokácia pamäte pre matice A, B, C
  Matrix A, B, C;
  A.width = A.height = A.stride = MAT_SIZE;
  B.width = B.height = B.stride = MAT_SIZE;
  C.width = C.height = C.stride = MAT_SIZE;
  A.elements = (float*) malloc(A.height * A.width * sizeof(float));
  B.elements = (float*) malloc(B.height * B.width * sizeof(float));
  C.elements = (float*) malloc(C.height * C.width * sizeof(float));

  //inicializácia matíc A, B a nulovanie C
  int i,j;
  for(i = 0; i < MAT_SIZE; i++){
    for(j = 0; j < MAT_SIZE; j++){
      A.elements[i * A.stride + j] = rand() % 10;
      B.elements[i * B.stride + j] = rand() % 10;
      C.elements[i * C.stride + j] = 0;
    }
  }
  
  //volanie násobenia matíc
  MatMul(A, B, C);

  for(i = 0; i < MAT_SIZE; i++){
    for(j = 0; j < MAT_SIZE; j++){
      printf("%f\t", A.elements[i * A.stride + j]);
    }
    printf("\n");
  }
  printf("\n");
  for(i = 0; i < MAT_SIZE; i++){
    for(j = 0; j < MAT_SIZE; j++){
      printf("%f\t", B.elements[i * A.stride + j]);
    }
    printf("\n");
  }
  printf("\n");
  for(i = 0; i < MAT_SIZE; i++){
    for(j = 0; j < MAT_SIZE; j++){
      printf("%f\t", C.elements[i * A.stride + j]);
    }
    printf("\n");
  }


  //uvolnenie pamäte
  free((void*) A.elements);
  free((void*) B.elements);
  free((void*) C.elements);
  
  return(0);
}
