#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void identify()
{
  printf("blok: x=%d y=%d z=%d\tvlakno: x=%d y=%d z=%d\n", blockIdx.x, blockIdx.y, blockIdx.z, threadIdx.x, threadIdx.y, threadIdx.z);
  printf("griddim: x=%d y=%d z=%d\tblokdim: x=%d y=%d z=%d\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
}

int main(int argc, char* argv[])
{
  //definícia dimenzie gridu
  dim3 gs(2,2);
  //definícia dimenzie bloku
  dim3 bs(32);
  //volanie kernelu
  identify<<<gs, bs>>>();
  //synchronizácia - dokončenie kernelu
  hipDeviceSynchronize();

  return(0);
}

