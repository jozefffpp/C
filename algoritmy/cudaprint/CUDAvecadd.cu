#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#define MAX 1024

__global__ void vecAdd(float* A, float* B, float* C)
{
  C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

int main(int argc, char* argv[])
{
  //ukazovatele na vektory v hostiteľskej pamäti
  float *hA, *hB, *hC;
  //ukazovatele na vektory v pamäti zariadenie GPGPU
  float *dA, *dB, *dC;

  //alokácia hostiteľskej pamäte
  hA = (float*) malloc(sizeof(float) * MAX);
  hB = (float*) malloc(sizeof(float) * MAX);
  hC = (float*) malloc(sizeof(float) * MAX);

  //inicializácia vstupných a výstupného vektoru
  for(int i = 0; i < MAX; i++) {
    hA[i] = hB[i] = i;
    hC[i] = 0;
  }

  //alokácaía pamäte zariadenia GPGPU
  hipMalloc(&dA, sizeof(float) * MAX);
  hipMalloc(&dB, sizeof(float) * MAX);
  hipMalloc(&dC, sizeof(float) * MAX);

  //kopírovanie údajov do GPGPU pamäte
  hipMemcpy(dA, hA, sizeof(float) * MAX, hipMemcpyHostToDevice);
  hipMemcpy(dB, hB, sizeof(float) * MAX, hipMemcpyHostToDevice);

 //definícia dimenzie gridu
  dim3 gs(1);
  //definícia dimenzie bloku
  dim3 bs(MAX);
  //volanie kernelu
  vecAdd<<<gs, bs>>>(dA, dB, dC);
  //synchronizácia - dokončenie kernelu
  hipDeviceSynchronize();

  //kopírovanie údajov z GPGPU pamäte
  hipMemcpy(hC, dC, sizeof(float) * MAX, hipMemcpyDeviceToHost);

  //uvolnenie pamäte zariadenia GPGPU
  hipFree(dA);
  hipFree(dB);
  hipFree(dC);

  //výpis výsledkov
  for(int i = 0; i < MAX; i++) {
    printf("%f ", hC[i]);
  }

  //uvolnenie hostiteľskej pamäte
  free(hA);
  free(hB);
  free(hC);

  return(0);
}

